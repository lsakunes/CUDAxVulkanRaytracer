#define GLFW_INCLUDE_VULKAN
#ifdef _WIN64
#include <aclapi.h>
#include <dxgi1_2.h>
#include <windows.h>
#include <VersionHelpers.h>
#define _USE_MATH_DEFINES
#endif

#include <GLFW/glfw3.h>
#include <vulkan/vulkan.h>
#ifdef _WIN64
#include <vulkan/vulkan_win32.h>
#endif

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_image.h>
#include <hip/hip_vector_types.h>

#include "linmath.h"

#define WIDTH 800
#define HEIGHT 600

const int MAX_FRAMES = 4;

const std::vector<const char*> validationLayers = {
    "VK_LAYER_KHRONOS_validation" };

#ifdef NDEBUG
const bool enableValidationLayers = true;
#else
const bool enableValidationLayers = false;
#endif

std::string execution_path;

VkResult CreateDebugUtilsMessengerEXT(
    VkInstance instance, const VkDebugUtilsMessengerCreateInfoEXT* pCreateInfo,
    const VkAllocationCallbacks* pAllocator,
    VkDebugUtilsMessengerEXT* pDebugMessenger) {
    auto func = (PFN_vkCreateDebugUtilsMessengerEXT)vkGetInstanceProcAddr(
        instance, "vkCreateDebugUtilsMessengerEXT");
    if (func != nullptr) {
        return func(instance, pCreateInfo, pAllocator, pDebugMessenger);
    }
    else {
        return VK_ERROR_EXTENSION_NOT_PRESENT;
    }
};

const std::vector<const char*> deviceExtensions = {
    VK_KHR_SWAPCHAIN_EXTENSION_NAME,
    VK_KHR_EXTERNAL_MEMORY_EXTENSION_NAME,
    VK_KHR_EXTERNAL_SEMAPHORE_EXTENSION_NAME,
#ifdef _WIN64
    VK_KHR_EXTERNAL_MEMORY_WIN32_EXTENSION_NAME,
    VK_KHR_EXTERNAL_SEMAPHORE_WIN32_EXTENSION_NAME,
#else
    VK_KHR_EXTERNAL_MEMORY_FD_EXTENSION_NAME,
    VK_KHR_EXTERNAL_SEMAPHORE_FD_EXTENSION_NAME,
#endif
};

#ifdef _WIN64
class WindowsSecurityAttributes {
protected:
    SECURITY_ATTRIBUTES m_winSecurityAttributes;
    PSECURITY_DESCRIPTOR m_winPSecurityDescriptor;

public:
    WindowsSecurityAttributes();
    SECURITY_ATTRIBUTES* operator&();
    ~WindowsSecurityAttributes();
};

WindowsSecurityAttributes::WindowsSecurityAttributes() {
    m_winPSecurityDescriptor = (PSECURITY_DESCRIPTOR)calloc(
        1, SECURITY_DESCRIPTOR_MIN_LENGTH + 2 * sizeof(void**));

    PSID* ppSID =
        (PSID*)((PBYTE)m_winPSecurityDescriptor + SECURITY_DESCRIPTOR_MIN_LENGTH);
    PACL* ppACL = (PACL*)((PBYTE)ppSID + sizeof(PSID*));

    InitializeSecurityDescriptor(m_winPSecurityDescriptor,
        SECURITY_DESCRIPTOR_REVISION);

    SID_IDENTIFIER_AUTHORITY sidIdentifierAuthority =
        SECURITY_WORLD_SID_AUTHORITY;
    AllocateAndInitializeSid(&sidIdentifierAuthority, 1, SECURITY_WORLD_RID, 0, 0,
        0, 0, 0, 0, 0, ppSID);

    EXPLICIT_ACCESS explicitAccess;
    ZeroMemory(&explicitAccess, sizeof(EXPLICIT_ACCESS));
    explicitAccess.grfAccessPermissions =
        STANDARD_RIGHTS_ALL | SPECIFIC_RIGHTS_ALL;
    explicitAccess.grfAccessMode = SET_ACCESS;
    explicitAccess.grfInheritance = INHERIT_ONLY;
    explicitAccess.Trustee.TrusteeForm = TRUSTEE_IS_SID;
    explicitAccess.Trustee.TrusteeType = TRUSTEE_IS_WELL_KNOWN_GROUP;
    explicitAccess.Trustee.ptstrName = (LPTSTR)*ppSID;

    SetEntriesInAcl(1, &explicitAccess, NULL, ppACL);

    SetSecurityDescriptorDacl(m_winPSecurityDescriptor, TRUE, *ppACL, FALSE);

    m_winSecurityAttributes.nLength = sizeof(m_winSecurityAttributes);
    m_winSecurityAttributes.lpSecurityDescriptor = m_winPSecurityDescriptor;
    m_winSecurityAttributes.bInheritHandle = TRUE;
}

SECURITY_ATTRIBUTES* WindowsSecurityAttributes::operator&() {
    return &m_winSecurityAttributes;
}

WindowsSecurityAttributes::~WindowsSecurityAttributes() {
    PSID* ppSID =
        (PSID*)((PBYTE)m_winPSecurityDescriptor + SECURITY_DESCRIPTOR_MIN_LENGTH);
    PACL* ppACL = (PACL*)((PBYTE)ppSID + sizeof(PSID*));

    if (*ppSID) {
        FreeSid(*ppSID);
    }
    if (*ppACL) {
        LocalFree(*ppACL);
    }
    free(m_winPSecurityDescriptor);
}
#endif

void DestroyDebugUtilsMessengerEXT(VkInstance instance,
    VkDebugUtilsMessengerEXT debugMessenger,
    const VkAllocationCallbacks* pAllocator) {
    auto func = (PFN_vkDestroyDebugUtilsMessengerEXT)vkGetInstanceProcAddr(
        instance, "vkDestroyDebugUtilsMessengerEXT");
    if (func != nullptr) {
        func(instance, debugMessenger, pAllocator);
    }
}

struct QueueFamilyIndices {
    int graphicsFamily = -1;
    int presentFamily = -1;

    bool isComplete() { return graphicsFamily >= 0 && presentFamily >= 0; }
};